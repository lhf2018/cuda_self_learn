
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>


#define  N 65535

__global__ void vector_add_gpu(int *a, int *b, int *c, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // 获取线程索引
    const int t_n = gridDim.x * blockDim.x; // 跳步的步长，所有线程的数量

    while (tid < n)
    {
        c[tid] = a[tid] + b[tid];
        tid += t_n;
    }
}

int main() {
    clock_t start,end;//数据类型是clock_t，需要头文件#include<time.h>
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    for (int i = 0; i < N; ++i) // 为数组a、b赋值
    {
        a[i] = i;
        b[i] = i * i;
    }
    start=clock();

    hipMalloc(&dev_a, sizeof(int) * N);
    hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice);

    hipMalloc(&dev_b, sizeof(int) * N);
    hipMemcpy(dev_b, b, sizeof(int) * N, hipMemcpyHostToDevice);

    hipMalloc(&dev_c, sizeof(int) * N);
    hipMemcpy(dev_c, c, sizeof(int) * N, hipMemcpyHostToDevice);

    vector_add_gpu<<<100, 200>>>(dev_a, dev_b, dev_c, N);
    hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost);

    end=clock();

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);


    for(int i=0; i<N; ++i)
    {
        printf("%d + %d = %d \n", a[i], b[i], c[i]);
    }

    std::cout<<"running time is: "<<(double)(end-start)/CLOCKS_PER_SEC<<std::endl;
    return 0;

}